#include "hip/hip_runtime.h"
#include "CudaSurfaceObjectWriteTestRenderer.h"

#include "hip/hip_runtime.h"

#include ""
#include "cuda/std/cmath"

#include "math_functions.h"

using namespace b3d::renderer;

__global__ auto writeVertexBuffer(hipSurfaceObject_t surface, unsigned int width, unsigned int height) -> void
{
	// gridDim: This variable is of type dim3 and contains the dimensions of the grid.
	// blockIdx: This variable is of type uint3 and contains the block index within the grid.
	// blockDim: This variable is of type dim3 and contains the dimensions of the block.
	// threadIdx: This variable is of type uint3 and contains the thread index within the block.

	const auto x = min(blockIdx.x * blockDim.x + threadIdx.x, width - 1);
	const auto y = min(blockIdx.y * blockDim.y + threadIdx.y, height - 1);

	auto val = uint32_t{ 0xFFFFFFFF };
	
	if (x + y == 0)
	{
		// printf("Hello from global thread 0\n");
	}
	surf2Dwrite(val, surface, x * sizeof(uint32_t), y);
}

auto CudaSurfaceObjectWriteTestRenderer::onRender() -> void
{
	auto renderTargetFeatureParams = renderTargetFeature_->getParamsData();

	const auto fbSize = dim3{ renderTargetFeatureParams.colorRT.extent.width,
							   renderTargetFeatureParams.colorRT.extent.height,
							};
	auto cudaRet = hipSuccess;
	// Execute Kernel
	{
		const auto gridDimXAdd = fbSize.x % 32 == 0 ? 0 : 1; 
		const auto gridDimYAdd = fbSize.y % 32 == 0 ? 0 : 1; 
		auto gridDim =
			dim3{ fbSize.x / 32 + gridDimXAdd, fbSize.y / 32 + gridDimYAdd };
		auto blockDim = dim3{ 32, 32 };
		writeVertexBuffer<<<gridDim, blockDim>>>(renderTargetFeatureParams.colorRT.surfaces[0].surface, fbSize.x, fbSize.y);
		hipDeviceSynchronize();
		cudaRet = hipGetLastError();
	}
}

auto CudaSurfaceObjectWriteTestRenderer::onInitialize() -> void
{
	
}

auto CudaSurfaceObjectWriteTestRenderer::onDeinitialize() -> void
{
	
}

auto CudaSurfaceObjectWriteTestRenderer::onGui() -> void
{
	
}
